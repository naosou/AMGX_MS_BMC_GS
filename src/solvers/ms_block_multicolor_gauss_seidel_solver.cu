#include "hip/hip_runtime.h"

#include <string.h>
#include <cutil.h>
#include <miscmath.h>
#include <amgx_cusparse.h>
#include <thrust/copy.h>
#include <solvers/ms_block_multicolor_gauss_seidel_solver.h>
#include <solvers/block_common_solver.h>
#include <gaussian_elimination.h>
#include <basic_types.h>
#include <util.h>
#include <texture.h>
#include <device_properties.h>
#include <stream.h>

#include "sm_utils.inl"

namespace amgx {

template <class T_Config>
MSBlockMultiColorGSSolver<T_Config>::MSBlockMultiColorGSSolver(AMG_Config &cfg, const std::string &cfg_scope)
{
    this->alpha = cfg.template getParameter<int>("alpha", cfg_scope);
    this->block_size = cfg.template getParameter<int>("block_size", cfg_scope);
    this->warp_size = cfg.template getParameter<int>("warp_size", cfg_scope);
    this->omega = cfg.template getParameter<ValueType>("omega", cfg_scope);
}

__global__ void ms_bmc_jacobi_kernel(const int *row_ptr, const int *col_ind, const float *values,
                                     const float *b, const float *x_old, float *x_new,
                                     const int *row_ids, int block_size, int alpha, float omega)
{
    int warp_id = blockIdx.x;
    int lane_id = threadIdx.x % 32;

    extern __shared__ float x_shared[];

    for (int rep = 0; rep < alpha; ++rep)
    {
        for (int i = lane_id; i < block_size; i += 32)
        {
            int row = row_ids[warp_id * block_size + i];
            float diag = 0.0f;
            float sum = 0.0f;

            for (int jj = row_ptr[row]; jj < row_ptr[row + 1]; ++jj)
            {
                int col = col_ind[jj];
                float val = __ldg(&values[jj]);
                if (col == row)
                    diag = val;
                else
                    sum += val * __ldg(&x_old[col]);
            }

            float gs_update = (b[row] - sum) / diag;
            x_shared[i] = fmaf(omega, gs_update - x_old[row], x_old[row]);
        }
        __syncthreads();

        for (int i = lane_id; i < block_size; i += 32)
        {
            int row = row_ids[warp_id * block_size + i];
            x_new[row] = x_shared[i];
        }
        __syncthreads();

        float *tmp = (float *)x_old;
        x_old = x_new;
        x_new = tmp;
    }
}

template <class T_Config>
bool MSBlockMultiColorGSSolver<T_Config>::solve_iteration(ValueType &b, ValueType &x, bool xIsZero)
{
    using IndexType = typename T_Config::IndPrec;
    using Value = typename T_Config::VecPrec;

    const Matrix<T_Config> &A = *(this->matrix);
    const int num_rows = A.get_num_rows();

    std::vector<int> row_ids;
    int num_blocks = (num_rows + block_size - 1) / block_size;

    for (int b = 0; b < num_blocks; ++b)
    {
        for (int i = 0; i < block_size; ++i)
        {
            int row = b * block_size + i;
            row_ids.push_back(row < num_rows ? row : 0);
        }
    }

    int *d_row_ptr, *d_col_ind, *d_row_ids;
    float *d_vals, *d_x0, *d_x1, *d_b;

    hipMalloc(&d_row_ptr, sizeof(int) * (num_rows + 1));
    hipMalloc(&d_col_ind, sizeof(int) * A.get_num_nz());
    hipMalloc(&d_vals, sizeof(float) * A.get_num_nz());
    hipMalloc(&d_x0, sizeof(float) * num_rows);
    hipMalloc(&d_x1, sizeof(float) * num_rows);
    hipMalloc(&d_b, sizeof(float) * num_rows);
    hipMalloc(&d_row_ids, sizeof(int) * row_ids.size());

    hipMemcpy(d_row_ptr, A.row_offsets.raw(), sizeof(int) * (num_rows + 1), hipMemcpyHostToDevice);
    hipMemcpy(d_col_ind, A.col_indices.raw(), sizeof(int) * A.get_num_nz(), hipMemcpyHostToDevice);
    hipMemcpy(d_vals, A.values.raw(), sizeof(float) * A.get_num_nz(), hipMemcpyHostToDevice);
    hipMemcpy(d_x0, x.raw(), sizeof(float) * num_rows, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b.raw(), sizeof(float) * num_rows, hipMemcpyHostToDevice);
    hipMemcpy(d_row_ids, row_ids.data(), sizeof(int) * row_ids.size(), hipMemcpyHostToDevice);

    dim3 grid(num_blocks);
    dim3 block(warp_size);
    size_t shmem = sizeof(float) * block_size;

    ms_bmc_jacobi_kernel<<<grid, block, shmem>>>(d_row_ptr, d_col_ind, d_vals, d_b, d_x0, d_x1, d_row_ids, block_size, alpha, omega);

    hipMemcpy(x.raw(), d_x0, sizeof(float) * num_rows, hipMemcpyDeviceToHost);

    hipFree(d_row_ptr);
    hipFree(d_col_ind);
    hipFree(d_vals);
    hipFree(d_x0);
    hipFree(d_x1);
    hipFree(d_b);
    hipFree(d_row_ids);

    return true;
}

template <class T_Config>
void MSBlockMultiColorGSSolver<T_Config>::print_solver_parameters() const
{
    std::cout << "MS-BMC-GS solver parameters:" << std::endl;
    std::cout << "  alpha: " << alpha << std::endl;
    std::cout << "  block_size: " << block_size << std::endl;
    std::cout << "  warp_size: " << warp_size << std::endl;
    std::cout << "  omega: " << omega << std::endl;
}

#define AMGX_CASE_LINE(CASE) template class MSBlockMultiColorGSSolver<CASE>;
AMGX_FORALL_BUILDS(AMGX_CASE_LINE)
#undef AMGX_CASE_LINE

#define AMGX_DECLARE_FACTORY(CASE) \ 
    template void MSBlockMultiColorGSSolver<CASE>::registerFactory();

} // namespace amgx
